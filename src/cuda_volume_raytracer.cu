#include "hip/hip_runtime.h"
/*
Copyright (c) 2019 Paul Stahr

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <cinttypes>
#include <vector>
#include <limits>
#include <iostream>
#include <stdexcept>
#include <numeric>
#include <algorithm>
#include <cassert>
#include <sstream>
#include <omp.h>
#include <cstdio>
#include <thread>
#include "types.h"
#include "tuple_math.h"
#include "tuple_io.h"

#include "cuda_volume_raytracer.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        std::stringstream ss;
        ss << hipGetErrorString( err ) << " in " << file << " at line " << line << " (" << err << ")";
        throw std::runtime_error(ss.str());
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static int inited = 0;

template <class T, T V>
struct template_constant
{
    template_constant(){};    
    constexpr operator T() const { return V; }
    template <typename W>
    constexpr T operator ()(W const &) const{return V;}
    template <typename W, typename X>
    constexpr T operator ()(W const &, X const &) const{return V;}
};

bool init()
{
    int count = 0;
    if (hipSuccess != hipGetDeviceCount(&count))
    {
        std::cerr << "Failed to get device count, no cuda availible" << std::endl;
        return false;
    }
    std::cout << "Device Count" << count << std::endl;
    if (!inited)
    {
        //HANDLE_ERROR(hipSetDevice(1));
        //HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        inited = count;
        return true;
    }
    return false;
}

template <uint8_t dim, typename DirType>
struct raydata_t{
    cuda_tuple<pos_t,dim> _position;
    cuda_tuple<DirType,dim> _direction;
    brightness_t _remaining_brightness;
    uint32_t _iterations;
};

__host__ __device__ cuda_tuple<int32_t, 2> interpolate(
    cuda_tuple<diff_t, 2> *diff_interleaved,
    uint32_t bounds1,
    cuda_tuple<uint32_t,2> pos)
{
    cuda_tuple<int32_t,2> values[4];
    diff_interleaved += (pos.x >> 16) * bounds1 + (pos.y >> 16);
    
    for (uint8_t i = 0; i < 4; ++i)
    {
        values[i] = make_struct<int32_t,2>()(diff_interleaved[((i >> 1) & 1) * bounds1 + (i & 1)]);
        values[i] *= 0x10000;
    }

    uint32_t multr = (pos.x & 0xFFFF) * 0x100;
    uint32_t multl = 0x1000000 - multr;

    for (uint8_t i = 0; i < 2; ++i)
    {
        add<2>(values[i],values[i+2],multl,multr);
    }
    multr = (pos.y & 0xFFFF) * 0x100;
    multl = 0x1000000 - multr;

    add<2>(values[0],values[1],multl,multr);
    return values[0];
 }
 
__host__ __device__ cuda_tuple<int32_t,3> interpolate(
    cuda_tuple<diff_t,4> *diff_interleaved,
    uint32_t bounds1,
    uint32_t bounds2,
    cuda_tuple<uint32_t,3> pos)
{
    diff_interleaved += ((pos.x >> 16) * bounds1 + (pos.y >> 16)) * bounds2 + (pos.z >> 16);
    cuda_tuple<int32_t,3> values[8];

    for (uint8_t i = 0; i < 8; ++i)
    {
        values[i] = make_struct<int32_t,3>()(diff_interleaved[(((i >> 2)) * bounds1 + ((i >> 1) & 1)) * bounds2 + (i & 1)]);
        values[i] *= 0x10000;
    }
    
    uint32_t multr = (pos.x & 0xFFFF) * 0x100;
    uint32_t multl = 0x1000000 - multr;
    for (uint8_t i = 0; i < 4; ++i)
    {
        add<3>(values[i],values[i+4],multl,multr);
    }
    multr = (pos.y & 0xFFFF) * 0x100;
    multl = 0x1000000 - multr;
    for (uint8_t i = 0; i < 2; ++i)
    {
        add<3>(values[i],values[i+2],multl,multr);
    }
    multr = (pos.z & 0xFFFF) * 0x100;
    multl = 0x1000000 - multr;
    add<3>(values[0],values[1],multl,multr);
    return values[0];
}

/*template <uint8_t dim, typename T>
inline __host__ __device__ cuda_tuple<float, dim> interpolatef(
    T *diff_interleaved,
    uint16_t bounds1,
    uint16_t bounds2,
    cuda_tuple<pos_t,3> pos)
{
    diff_interleaved += ((pos.x >> 16) * static_cast<uint32_t>(bounds1) + (pos.y >> 16)) * static_cast<uint32_t>(bounds2) + (pos.z >> 16);
    cuda_tuple<float,dim> values[8];
    
    for (uint8_t i = 0; i < 8; ++i)
    {
        values[i] = make_struct<float,dim>()(diff_interleaved[((i >> 2) * static_cast<uint32_t>(bounds1) + ((i >> 1) & 1)) * static_cast<uint32_t>(bounds2) + (i & 1)]);
    }

    float multr = pos.x & 0xFFFF;
    float multl = 0x10000 - (pos.x & 0xFFFF);
    for (uint8_t i = 0; i < 4; ++i)
    {
        add<dim>(values[i],values[i+4],multl,multr);
    }
    multr = pos.y & 0xFFFF;
    multl = 0x10000 - (pos.x & 0xFFFF);
    for (uint8_t i = 0; i < 2; ++i)
    {
        add<dim>(values[i],values[i+2],multl,multr);
    }
    multr = pos.z & 0xFFFF;
    multl = 0x10000 - (pos.x & 0xFFFF);
    add<dim>(values[0],values[1],multl,multr);
    values[0] /= 0x1000000000000p0f;
    return values[0];
}*/

template <uint8_t dim>
inline __host__ __device__ size_t get_index(cuda_tuple<uint16_t, dim> bounds, cuda_tuple<uint32_t, dim> pos)
{
    return 0;
}

template <>
inline __host__ __device__ size_t get_index(cuda_tuple<uint16_t, 2> bounds, cuda_tuple<uint32_t, 2> pos)
{
    return (pos.x >> 16) * static_cast<uint32_t>(bounds.y) + (pos.y >> 16);
}

template <>
inline __host__ __device__ size_t get_index(cuda_tuple<uint16_t, 3> bounds, cuda_tuple<uint32_t, 3> pos)
{
    return ((pos.x >> 16) * static_cast<uint32_t>(bounds.y) + (pos.y >> 16)) * static_cast<uint32_t>(bounds.z) + (pos.z >> 16);
}

template <uint8_t T>
struct type_uint8_t{};

template <uint8_t dim, uint8_t dimtuple, typename T>
inline __host__ __device__ cuda_tuple<float, dimtuple> interpolatef(
    cuda_tuple<T, dimtuple> *diff_interleaved,
    cuda_tuple<uint16_t,dim> bounds,
    cuda_tuple<pos_t,dim> pos,
    type_uint8_t<dimtuple> td);//{return cuda_tuple<float, dimtuple>();}  

template <uint8_t dimtuple, typename T>
inline __host__ __device__ cuda_tuple<float, dimtuple> interpolatef(
    cuda_tuple<T, dimtuple> *diff_interleaved,
    cuda_tuple<uint16_t,3> bounds,
    cuda_tuple<pos_t,3> pos,
    type_uint8_t<dimtuple> td)
{
                        
    diff_interleaved += get_index(bounds, pos);
    cuda_tuple<float,dimtuple> values[8];
    
    for (uint8_t i = 0; i < 8; ++i)
    {
        values[i] = make_struct<float,dimtuple>()(diff_interleaved[((i >> 2) * static_cast<uint32_t>(bounds.y) + ((i >> 1) & 1)) * static_cast<uint32_t>(bounds.z) + (i & 1)]);
    }
    float multr = pos.x & 0xFFFF;
    float multl = 0x10000 - multr;
    for (uint8_t i = 0; i < 4; ++i)
    {
        add<dimtuple>(values[i],values[i+4],multl,multr);
    }
    multr = pos.y & 0xFFFF;
    multl = 0x10000 - multr;
    for (uint8_t i = 0; i < 2; ++i)
    {
        add<dimtuple>(values[i],values[i+2],multl,multr);
    }
    multr = pos.z & 0xFFFF;
    multl = 0x10000 - multr;
    add<dimtuple>(values[0],values[1],multl,multr);
    values[0] /= 0x1000000000000p0f;
    return values[0];
}

template <uint8_t dimtuple, typename T>
inline __host__ __device__ cuda_tuple<float, dimtuple> interpolatef(
    cuda_tuple<T, dimtuple> *diff_interleaved,
    cuda_tuple<uint16_t,2> bounds,
    cuda_tuple<pos_t,2> pos,
    type_uint8_t<dimtuple> td)
{
    diff_interleaved += get_index(bounds, pos);
    cuda_tuple<float,dimtuple> values[4];
    
    for (uint8_t i = 0; i < 4; ++i)
    {
        values[i] = make_struct<float,dimtuple>()(diff_interleaved[((i >> 1) & 1) * static_cast<uint32_t>(bounds.y) + (i & 1)]);
    }

    float multr = pos.x & 0xFFFF;
    float multl = 0x10000 - multr;
    for (uint8_t i = 0; i < 2; ++i)
    {
        add<dimtuple>(values[i],values[i+2],multl,multr);
    }
    multr = pos.y & 0xFFFF;
    multl = 0x10000 - multr;
    add<dimtuple>(values[0],values[1],multl,multr);
    values[0] /= 0x100000000p0f;
    return values[0];
}

/*template <typename P, typename T, typename B>
__global__ void trace_ray(
    cuda_tuple<diff_t,2> *diff_interleaved,
    T translucency,
    cuda_tuple<uint32_t,2> bounds,
    raydata_t<2, dir_t> *raydata,
    P path,
    B minimum_brightness,
    uint32_t n)
{
    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        raydata += i;
        cuda_tuple<pos_t,2> pos = raydata->_position;
        cuda_tuple<int32_t,2> direction = make_struct<int32_t,2>()(raydata->_direction);
        uint32_t iterations = raydata->_iterations;
        path += iterations * i;
        direction *= 0x10000;
        
        B brightness = 0xFFFFFFFF;
        while (iterations -- > 0 && (pos >> 16) < bounds - 1)
        {
            if (translucency)
            {
                //printf("%u %u\n",pos.x>>16,pos.y>>16);
                brightness -= min(brightness, 0xFFFFFFFF-translucency[(pos.x >> 16) * bounds.y + (pos.y >> 16)]);
                if (brightness < minimum_brightness)
                {
                    break;
                }
            }
            cuda_tuple<int32_t,2> erg = interpolate(diff_interleaved, bounds.y, pos);
            erg >>= 12;
            direction += erg;
            pos += direction >> 16;
            path[iterations] = pos;
        }
        if (path)
        {
            ++iterations;
            while (iterations --> 0)
            {
                path[iterations] = pos;
            }
        }
        direction /= 0x10000;
        raydata->_position = pos;
        raydata->_direction = make_struct<int16_t,2>()(direction);
        raydata->_remaining_brightness = brightness;
        raydata->_iterations = iterations;
    }
}

template <typename P, typename T, typename B>
__global__ void trace_ray(
    cuda_tuple<diff_t,4> *diff_interleaved,
    T translucency,
    cuda_tuple<uint32_t,3> bounds,
    raydata_t<3, dir_t> *raydata,
    P path,
    B minimum_brightness,
    uint16_t n)
{
    uint16_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        raydata += i;
        path += iterations * 3 * i;
        cuda_tuple<uint32_t,3> pos = raydata->_position;
        cuda_tuple<int32_t,3> direction = make_struct<int32_t,3>()(raydata->_direction);
        uint32_t iterations = raydata._iterations;
        direction *= 0x10000;

        B brightness = 0xFFFFFFFF;
        while (iterations -- > 0 && (pos >> 16) < bounds - 1)
        {
            if (translucency)
            {
                brightness -= min(brightness, 0xFFFFFFFF-translucency[((pos.x >> 16) * bounds.y + (pos.y >> 16)) * bounds.z + (pos.z >> 16)]);
                if (brightness < minimum_brightness)
                {
                    break;
                }
            }
            cuda_tuple<int32_t,3> erg = interpolate(diff_interleaved, bounds.y, bounds.z, pos);
            erg >>= 12;
            direction += erg;
            pos += direction >> 16;
            path[iterations] = pos;
        }
        if (path)
        {
            ++iterations;
            while (iterations --> 0)
            {
                path[iterations] = pos;
            }
        }
        direction /= 0x10000;
        raydata->_position = pos;
        raydata->_direction = make_struct<dir_t,3>()(direction);
        raydata->_remaining_brightness = brightness;
        raydata->_iterations = iterations;
    }
}*/

class DummyObject
{
public:
    inline __host__ __device__ DummyObject(){}
    
    template <typename T>
    inline __host__ __device__ DummyObject(T t){}
    
    template <typename T>
    inline __host__ __device__ T& operator=(T&& other) noexcept{return other;}
    
    template <typename T>
    inline __host__ __device__ operator T() const{return T();}
    
    template <typename T>
    inline __host__ __device__ DummyObject operator-=(T value){return *this;}
    
    template <typename T>
    inline __host__ __device__ DummyObject operator<(T value){return false;}
    
    
};

template <typename T>
inline __host__ __device__ DummyObject operator -(T a, DummyObject b){return DummyObject();}

//template<class T>
//void operator=(T& value, const DummyObject& v)
//{}

class DummyArray{
    public:
    inline __host__ __device__ DummyObject const operator [](size_t index) const{return DummyObject();}

    inline __host__ __device__ DummyObject operator [](size_t index) {return DummyObject();}
    
    inline __host__ __device__ void operator +=(size_t index) {}

    inline __host__ __device__ DummyArray operator +(size_t index) {return DummyArray();}

    inline __host__ __device__ operator bool() const{return false;}
};

template <typename P, typename T, typename B, typename DiffType, typename DirType, uint8_t dim>
inline __host__ __device__ void trace_ray_function(
    cuda_tuple<DiffType,dim + 1>  *diff_interleaved,
    T translucency,/*translucency_t*/
    cuda_tuple<uint16_t,dim> bounds,
    cuda_tuple<float,dim> invscale,
    raydata_t<dim, DirType> *raydata,
    P path, /*cuda_tuple<pos_t,dim>*/    
    B minimum_brightness)
{
    cuda_tuple<uint32_t,dim> pos = raydata->_position;
    cuda_tuple<float,dim> direction = make_struct<float,dim>()(raydata->_direction);
    uint32_t iterations = raydata->_iterations;
    if(std::is_same<dir_t, DirType>::value)
    {
        direction *= 0x100;
    }
    else
    {
        direction *= 0x10000;
    }
    B brightness = 0xFFFFFFFF;
    path[--iterations] = pos;

    while (iterations -- > 0 && make_struct<uint16_t, dim>()(pos >> 16) < bounds - 1)
    {
        if (translucency)
        {
            brightness -= min(static_cast<brightness_t>(brightness), static_cast<translucency_t>(0xFFFFFFFF-translucency[get_index(bounds, pos)]));
            if (brightness < minimum_brightness)
            {
                //printf("b=%u<%u=mb", brightness, minimum_brightness);
                break;
            }
        }
        cuda_tuple<float,dim + 1> interpolation = interpolatef(diff_interleaved, bounds, pos, type_uint8_t<dim + 1>());
        if (get(interpolation, dim) > 0)
        {
            break;
        }
        interpolation *= invscale;//TODO can be precalculated
        direction += interpolation;
        float ilen = 0x40000000p0f / dot(direction, direction);
        pos += __float2int_rn2(direction * invscale * ilen);
        path[iterations] = pos;
    }
    if (!std::is_same<P,DummyArray>::value)
    {
        ++iterations;
        while (iterations --> 0)
        {
            path[iterations] = pos;
        }
    }
    if(std::is_same<dir_t, DirType>::value)
    {
        direction /= 0x100;
        raydata->_direction = make_struct<DirType,dim>()(__float2int_rn2(direction));
    }
    else
    {
        direction /= 0x10000;
        raydata->_direction = make_struct<DirType,dim>()(direction);
    }
    raydata->_position = pos;
    if (translucency)
    {
        raydata->_remaining_brightness = brightness;
    }
    raydata->_iterations = iterations;
}

template <typename P, typename T, typename B, typename DiffType, typename DirType, uint8_t dim>
void trace_rays_cpu(
    DiffType *diff_interleaved,
    T translucency,
    cuda_tuple<uint16_t,dim> bounds,
    cuda_tuple<float,dim> invscale,
    raydata_t<dim, DirType> *raydata,
    P path,
    uint32_t iterations,
    B minimum_brightness,
    size_t blocksize,
    size_t num_threads)
{
    #pragma omp parallel for num_threads(num_threads) if(blocksize > 0x100)
    for (size_t i = 0; i < blocksize; ++i)
    {
        trace_ray_function(reinterpret_cast<cuda_tuple<DiffType,dim + 1>* >(diff_interleaved), translucency, bounds, invscale, raydata + i, path + iterations * i, minimum_brightness);
    }
}

template <typename P, typename T, typename B, typename DiffType, typename DirType, uint8_t dim>
__global__ void trace_rays_gpu(
    DiffType *diff_interleaved,
    T translucency,
    cuda_tuple<uint16_t,dim> bounds,
    cuda_tuple<float,dim> invscale,
    raydata_t<dim, DirType> *raydata,
    P path,
    uint32_t iterations,
    B minimum_brightness,
    uint16_t n)
{
    uint16_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        trace_ray_function(reinterpret_cast<cuda_tuple<DiffType,dim + 1>* >(diff_interleaved), translucency, bounds, invscale, raydata + i, path + iterations * i, minimum_brightness);
    }
}


template <typename U, typename T>
void interleave(U input, size_t num_rows, size_t num_cols, std::vector<T> & out)
{
    out.clear();
    std::cout << "reserve " << num_cols << '*' << num_rows << std::endl; 
    out.reserve(num_cols * num_rows);
    for (size_t i = 0; i < num_cols; ++i)
    {
        for (size_t j = 0; j < num_rows; ++j)
        {
            out.emplace_back(input[j][i]);
        }
    }
}

/*template <typename T>
void interleave(std::initializer_list<std::vector<T> > const & input, std::vector<T> & out)
{
    out.clear();
    size_t size = (*input.begin()).size();
    out.reserve(size * input.size());
    for (size_t i = 0; i < size; ++i)
    {
        for (std::vector<T> const & vec : input)
        {
            out.push_back(vec[i]);
        }
    }
}



template <typename T>
void interleave3(std::vector<std::vector<T> const * > input, std::vector<T> & out)
{
    out.clear();
    size_t size = (input[0])->size();
    out.reserve(size * input.size());
    for (size_t i = 0; i < size; ++i)
    {
        for (std::vector<T> const * vec : input)
        {
            out.emplace((*vec)[i]);
        }
    }
}

template <typename T, uint8_t dim>
void interleave2(std::array<std::vector<T> const *, dim > input, std::vector<T> & out)
{
    out.clear();
    size_t size = (*input.begin()).size();
    out.reserve(size * input.size());
    for (size_t i = 0; i < size; ++i)
    {
        for (std::vector<T> const & vec : *input)
        {
            out.emplace(vec[i]);
        }
    }
}
*/
template <int dim, typename DirType>
void fill_struct(
    std::vector<pos_t> const & start_position,
    std::vector<DirType> const & start_direction,
    size_t iterations,
    std::vector<raydata_t<dim, DirType> > & raydata)
{
    size_t num_rays = start_position.size() / dim;
    raydata.clear();
    raydata.reserve(num_rays);
    for (size_t i = 0; i < num_rays; ++i)
    {
        raydata.emplace_back();
        raydata_t<dim, DirType> & back = raydata.back();
        back._position = reinterpret_cast<cuda_tuple<pos_t, dim>const * >(start_position.data())[i];
        back._direction = reinterpret_cast<cuda_tuple<DirType, dim>const * >(start_direction.data())[i];
        
        back._remaining_brightness=std::numeric_limits<brightness_t>::max();
        back._iterations = iterations;
    } 
}

template <int dim, typename DirType>
void read_struct(
    std::vector<pos_t> & position,
    std::vector<DirType> & direction,
    std::vector<brightness_t> & remaining_light,
    std::vector<raydata_t<dim, DirType> > const & raydata)
{
    bool warn = false;
    for (size_t i = 0; i < raydata.size(); ++i)
    {
        raydata_t<dim, DirType> const & current = raydata[i];
        
        reinterpret_cast<cuda_tuple<pos_t, dim> * >(position.data())[i] = current._position;
        reinterpret_cast<cuda_tuple<DirType, dim> * >(direction.data())[i] = current._direction;
        remaining_light[i] = current._remaining_brightness;
        if (current._iterations == 0 || current._iterations == std::numeric_limits<uint32_t>::max())
        {
            warn = true;
        }
    }
    if (warn)
    {
        std::cout << "Warning, maximum iterations hitted" << std::endl;
    }
}
    
template <typename T>
size_t inline sizeofvec(std::vector<T> const & vec)
{
    //return vec.size()      * sizeof(decltype(vec)::value_type);
    return vec.size()      * sizeof(T);
}

/*void trace_rays_cu(
    std::vector<size_t> const & output_sizes,
    std::vector<diff_t> const & diffx,
    std::vector<diff_t> const & diffy,
    std::vector<translucency_t> const & translucency_cropped,
    std::vector<pos_t> const & start_position,
    std::vector<dir_t> const & start_direction,
    std::vector<pos_t> & end_position,
    std::vector<dir_t> & end_direction,
    std::vector<brightness_t> & remaining_light,
    std::vector<pos_t> & path,
    std::vector<float> const & invscale_vec,
    brightness_t minimum_brightness,
    uint32_t iterations,
    bool trace_paths,
    Options const & opt)
{    
    init();
    cuda_tuple<diff_t,2> *diff_interleaved_cuda;
    translucency_t *translucency_cuda;
    cuda_tuple<pos_t,2> *path_cuda = nullptr;
    raydata_t<2, dir_t> *raydata_cuda;
    size_t num_rays = start_position.size() / 2;
    if (trace_paths)
    {
        path.resize(iterations * 2 * num_rays);
    }
    
    std::vector<raydata_t<2, dir_t> > ray_data;
    fill_struct<2>(start_position, start_direction, iterations, ray_data);
    HANDLE_ERROR(hipMalloc(&raydata_cuda,sizeofvec(ray_data)));
    HANDLE_ERROR(hipMemcpyAsync(raydata_cuda,ray_data.data(),     sizeofvec(ray_data), hipMemcpyHostToDevice));
    std::vector<diff_t> diff_interleaved;
    interleave({diffy, diffx}, diff_interleaved);
    HANDLE_ERROR(hipMalloc(&diff_interleaved_cuda,diff_interleaved.size()      * sizeof(diff_t)));
    HANDLE_ERROR(hipMalloc(&translucency_cuda,    translucency_cropped.size()  * sizeof(translucency_t)));
    if (trace_paths)
    {
        HANDLE_ERROR(hipMalloc(&path_cuda,            path.size()                  * sizeof(pos_t)));
    }

    HANDLE_ERROR(hipMemcpyAsync(diff_interleaved_cuda,           diff_interleaved.data(),                diff_interleaved.size()                 * sizeof(diff_t), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpyAsync(translucency_cuda,    translucency_cropped.data(), translucency_cropped.size()  * sizeof(translucency_t), hipMemcpyHostToDevice));
    float2 invscale = make_float2(invscale_vec[0],invscale_vec[1]);

    size_t maximum_rays_per_kernel = 32768;
    size_t threads_per_block = 128;
    //size_t maximum_rays_per_kernel = 64;
    //size_t threads_per_block = 32;
    
    for (size_t i = 0; i < num_rays; i += maximum_rays_per_kernel)
    {
        size_t kernel_rays = std::min(maximum_rays_per_kernel, num_rays - i * maximum_rays_per_kernel);
        size_t block_count = (kernel_rays + threads_per_block - 1)/threads_per_block;

        size_t shared_mem = 0;
        if (path_cuda == nullptr)
        {
        trace_ray<<<block_count, threads_per_block, shared_mem>>>(
            diff_interleaved_cuda,
            translucency_cuda,
            make_struct<uint32_t,2>()(output_sizes[0],output_sizes[1]),
            raydata_cuda + i * maximum_rays_per_kernel,
            DummyArray(),
            iterations,
            minimum_brightness,
            kernel_rays);
        }
        else
        {
        trace_ray<<<block_count, threads_per_block, shared_mem>>>(
            diff_interleaved_cuda,
            translucency_cuda,
            make_struct<uint32_t,2>()(output_sizes[0],output_sizes[1]),
            raydata_cuda + i * maximum_rays_per_kernel,
            path_cuda + i * iterations,
            iterations,
            minimum_brightness,
            kernel_rays);
        }
    }

    HANDLE_ERROR(hipMemcpyAsync(ray_data.data(),    raydata_cuda,    sizeofvec(ray_data), hipMemcpyDeviceToHost));
    read_struct<2>(end_position, end_direction, remaining_light, ray_data);
    if (trace_paths)
    {
        HANDLE_ERROR(hipMemcpyAsync(path.data(),            path_cuda,            path.size()            * sizeof(pos_t),  hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipFree(diff_interleaved_cuda));
    HANDLE_ERROR(hipFree(raydata_cuda));
    HANDLE_ERROR(hipFree(translucency_cuda));
    if (trace_paths)
    {
        HANDLE_ERROR(hipFree(path_cuda));
    }
}*/

template <typename T>
std::vector<std::vector<T> const *> convert_to_references(std::vector<std::vector<T> > const & data)
{
    std::vector<std::vector<T> const *> res;
    res.reserve(data.size());
    for (std::vector<T> const & d : data)
    {
        res.push_back(&d);
    }
    return res;
}

template <typename DiffType>
TraceRaysCu<DiffType>::TraceRaysCu(
    std::vector<size_t> const & output_sizes_,
    std::vector<std::vector<DiffType> > const & diff_,
    std::vector<translucency_t> const & translucency_cropped_) : TraceRaysCu(output_sizes_, convert_to_references(diff_), translucency_cropped_)
{}

template <typename DiffType>
TraceRaysCu<DiffType>::TraceRaysCu(
        std::vector<size_t> const & bounds,
        std::vector<std::vector<DiffType> const * > const & diff,
        std::vector<translucency_t> const & translucency_cropped) : _translucency_cropped(translucency_cropped)
{
    init();
    uint8_t dim = bounds.size();
    _output_sizes.reserve(dim);
    std::copy(bounds.begin(), bounds.end(), std::back_inserter(_output_sizes));
    std::vector<DiffType> extra_component;
    extra_component.reserve(diff[0]->size());
    for (translucency_t tr : translucency_cropped)
    {
        extra_component.push_back((static_cast<int64_t>(0x7FFFFFFF)-(static_cast<int64_t>(tr)))/0x10000);
    }
    std::vector<typename std::vector<DiffType>::const_iterator> tmp;
    tmp.reserve(bounds.size() + 1);
    for (std::vector<DiffType> const * d : diff)
    {
        tmp.push_back(d->cbegin());
    }
    tmp.push_back(extra_component.cbegin()); 
    interleave(tmp, tmp.size(), diff[0]->size(), _diff_interleaved);
    _diff_interleaved_cuda.resize(inited);
    _translucency_cuda.resize(inited);
    for (size_t i = 0; i < inited; ++i)
    {
        HANDLE_ERROR(hipSetDevice(i));
        HANDLE_ERROR(hipMalloc(&_diff_interleaved_cuda[i],_diff_interleaved.size()      * sizeof(DiffType)));
        HANDLE_ERROR(hipMalloc(&_translucency_cuda[i],    _translucency_cropped.size()  * sizeof(translucency_t)));

        HANDLE_ERROR(hipMemcpyAsync(_diff_interleaved_cuda[i],_diff_interleaved.data(),     _diff_interleaved.size()      * sizeof(DiffType), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpyAsync(_translucency_cuda[i],    _translucency_cropped.data(), _translucency_cropped.size()  * sizeof(translucency_t), hipMemcpyHostToDevice));
    }
}
    
template <typename DiffType>
template <typename DirType>
void TraceRaysCu<DiffType>::trace_rays_cu(
    std::vector<pos_t> const & start_position,
    std::vector<DirType> const & start_direction,
    std::vector<pos_t> & end_position,
    std::vector<DirType> & end_direction,
    std::vector<brightness_t> & remaining_light,
    std::vector<pos_t> & path,
    std::vector<float> const & invscale_vec,
    brightness_t minimum_brightness,
    uint32_t iterations,
    bool trace_paths,
    Options const & opt)
    {
        if (_output_sizes.size() == 2)
        {
            trace_rays_cu_impl<DirType, 2>(start_position,
                start_direction,
                end_position,
                end_direction,
                remaining_light,
                path,
                invscale_vec,
                minimum_brightness,
                iterations,
                trace_paths,
                opt);
        }
        else if (_output_sizes.size() == 3)
        {
            trace_rays_cu_impl<DirType, 3>(start_position,
                start_direction,
                end_position,
                end_direction,
                remaining_light,
                path,
                invscale_vec,
                minimum_brightness,
                iterations,
                trace_paths,
                opt);
        }
        else
        {
            throw std::runtime_error("Illegal dimension");
        }
    }
    
template <typename DiffType>
template <typename DirType, uint8_t dim>
void TraceRaysCu<DiffType>::trace_rays_cu_impl(
    std::vector<pos_t>          const & start_position,
    std::vector<DirType>        const & start_direction,
    std::vector<pos_t> &        end_position,
    std::vector<DirType> &      end_direction,
    std::vector<brightness_t> & remaining_light,
    std::vector<pos_t> &        path,
    std::vector<float> const &  invscale_vec,
    brightness_t                minimum_brightness,
    uint32_t                    iterations,
    bool                        trace_paths,
    Options const &             opt)
{    
    size_t num_rays = start_position.size() / dim;
    if (trace_paths)
    {
        path.resize(iterations * dim * num_rays);
    }

    std::vector<raydata_t<dim, DirType> > ray_data;
    fill_struct<dim>(start_position, start_direction, iterations, ray_data);
    size_t maximum_rays_per_kernel = 0x8000;
    size_t threads_per_block = 0x80;
    //size_t maximum_rays_per_kernel = 64;
    //size_t threads_per_block = 32;
    size_t cuda_device_count = inited;
    if (num_rays <= 0x80)
    {
        cuda_device_count = 0;
    }
    cuda_device_count = std::min(cuda_device_count, (num_rays + maximum_rays_per_kernel - 1) / maximum_rays_per_kernel);
    std::vector<cuda_tuple<pos_t,dim> *> path_cuda(cuda_device_count);
    std::cout << "allocate " << cuda_device_count << std::endl;
    std::vector<raydata_t<dim, DirType>* > raydata_cuda(cuda_device_count);
    size_t cpu_device_count = omp_get_max_threads() - cuda_device_count;
    if (cuda_device_count != 0)
    {
        cpu_device_count = 0;
    }
    
    cuda_tuple<float,dim> invscale = make_struct<float,dim>()(invscale_vec.data());
    cuda_tuple<uint16_t, dim> output_sizes = make_struct<uint16_t, dim>()(_output_sizes.data());
    omp_set_nested(1);
    size_t count_cpu = 0;
    size_t count_gpu = 0;
    assert(diff_interleaved.size() == prod(output_sizes));
    size_t num_parallel = cuda_device_count + (cpu_device_count > 0);
    #pragma omp parallel for schedule(dynamic) num_threads(num_parallel) if (num_parallel > 1)
    for (size_t i = 0; i < num_rays; i += maximum_rays_per_kernel)
    {
        size_t num_kernel_rays = std::min(maximum_rays_per_kernel, num_rays - i);
        size_t block_count = (num_kernel_rays + threads_per_block - 1)/threads_per_block;
        if (opt._loglevel > 0)
        {
            std::cout << "iteration " << (i / maximum_rays_per_kernel) << " of " << (num_rays + maximum_rays_per_kernel - 1) / maximum_rays_per_kernel << std::endl;
        }
        size_t thread_num = omp_get_thread_num();
        if (thread_num < cuda_device_count)
        {
            if (raydata_cuda[thread_num] == nullptr)
            {
                HANDLE_ERROR(hipSetDevice(thread_num));
                HANDLE_ERROR(hipMalloc(&raydata_cuda[thread_num],std::min(ray_data.size(),maximum_rays_per_kernel)      * sizeof(raydata_t<dim, DirType>)));
                if (trace_paths)
                {
                    HANDLE_ERROR(hipMalloc(&path_cuda[thread_num],            std::min(ray_data.size(), maximum_rays_per_kernel) * iterations * dim * sizeof(pos_t)));
                }
            }
            std::cout << thread_num << " gpu: " <<  num_kernel_rays << std::endl;
            HANDLE_ERROR(hipMemcpyAsync(raydata_cuda[thread_num],ray_data.data() + i,     num_kernel_rays      * sizeof(raydata_t<dim, DirType>), hipMemcpyHostToDevice));
            if (trace_paths)
            {
                trace_rays_gpu<<<block_count, threads_per_block>>>(
                    _diff_interleaved_cuda[thread_num],
                    _translucency_cuda[thread_num],
                    output_sizes,
                    invscale,
                    raydata_cuda[thread_num],
                    path_cuda[thread_num],
                    iterations,
                    minimum_brightness,
                    num_kernel_rays);
            }
            else
            {
                trace_rays_gpu<<<block_count, threads_per_block>>>(
                    _diff_interleaved_cuda[thread_num],
                    DummyArray(),//_translucency_cuda,
                    output_sizes,
                    invscale,
                    raydata_cuda[thread_num],
                    DummyArray(),
                    iterations,
                    DummyObject(),//minimum_brightness,
                    num_kernel_rays);
            }
            HANDLE_ERROR(hipDeviceSynchronize());
            HANDLE_ERROR(hipMemcpyAsync(ray_data.data() + i,    raydata_cuda[thread_num],    num_kernel_rays   * sizeof(raydata_t<dim, DirType>), hipMemcpyDeviceToHost));
            if (trace_paths)
            {
                HANDLE_ERROR(hipMemcpyAsync(path.data() + i * dim * iterations,            path_cuda[thread_num],            path.size()            * sizeof(pos_t),  hipMemcpyDeviceToHost));
            }
            ++count_gpu;
        }
        else
        {
            std::cout << thread_num << " cpu: " <<  num_kernel_rays << std::endl;
            if (trace_paths)
            {
                trace_rays_cpu(
                    _diff_interleaved.data(),
                    DummyArray(),//_translucency_cropped.data(),
                    output_sizes,
                    invscale,
                    ray_data.data() + i,
                    reinterpret_cast<cuda_tuple<pos_t,dim>*>(path.data()) + i * iterations,
                    iterations,
                    DummyObject(),//minimum_brightness,
                    num_kernel_rays,
                    cpu_device_count);
            }else{
                trace_rays_cpu(
                    _diff_interleaved.data(),
                    DummyArray(),//_translucency_cropped.data(),
                    output_sizes,
                    invscale,
                    ray_data.data() + i,
                    DummyArray(),
                    iterations,
                    DummyObject(),//minimum_brightness,
                    num_kernel_rays,
                    cpu_device_count);
            }
            ++count_cpu;
        }
        //std::cout << ray_data[i]._position[0] << std::endl;
        //std::this_thread::sleep_for(std::chrono::milliseconds(10));
    }
    
    if (opt._loglevel > 0)
    {
        std::cout << "cpu: " << count_cpu << " gpu: " << count_gpu << std::endl;
    }
    read_struct<dim>(end_position, end_direction, remaining_light, ray_data);
    for (size_t i = 0; i < cuda_device_count; ++i)
    {
        HANDLE_ERROR(hipSetDevice(i));
        HANDLE_ERROR(hipFree(raydata_cuda[i]));
        if (trace_paths)
        {
            HANDLE_ERROR(hipFree(path_cuda[i]));
        }        
    }
    if (cuda_device_count != 0)
    {
        HANDLE_ERROR(hipDeviceSynchronize());
    }
}
    
template <typename DiffType>
TraceRaysCu<DiffType>::~TraceRaysCu()
{
    for (size_t i = 0; i < inited; ++i)
    {
        HANDLE_ERROR(hipSetDevice(i));
        HANDLE_ERROR(hipFree(_diff_interleaved_cuda[i]));
        HANDLE_ERROR(hipFree(_translucency_cuda[i]));
    }
    if (inited)
    {
        HANDLE_ERROR(hipDeviceSynchronize());
    }
}


template class TraceRaysCu<diff_t>;
template class TraceRaysCu<float>;

template void TraceRaysCu<diff_t>::trace_rays_cu<dir_t>(
        std::vector<pos_t> const & start_position,
        std::vector<dir_t> const & start_direction,
        std::vector<pos_t> & end_position,
        std::vector<dir_t> & end_direction,
        std::vector<brightness_t> & remaining_light,
        std::vector<pos_t> & path,
        std::vector<float> const & invscale_vec,
        brightness_t minimum_brightness,
        uint32_t iterations,
        bool trace_paths,
        Options const & opt);

template void TraceRaysCu<float>::trace_rays_cu<dir_t>(
        std::vector<pos_t> const & start_position,
        std::vector<dir_t> const & start_direction,
        std::vector<pos_t> & end_position,
        std::vector<dir_t> & end_direction,
        std::vector<brightness_t> & remaining_light,
        std::vector<pos_t> & path,
        std::vector<float> const & invscale_vec,
        brightness_t minimum_brightness,
        uint32_t iterations,
        bool trace_paths,
        Options const & opt);
        
template void TraceRaysCu<diff_t>::trace_rays_cu<float>(
        std::vector<pos_t> const & start_position,
        std::vector<float> const & start_direction,
        std::vector<pos_t> & end_position,
        std::vector<float> & end_direction,
        std::vector<brightness_t> & remaining_light,
        std::vector<pos_t> & path,
        std::vector<float> const & invscale_vec,
        brightness_t minimum_brightness,
        uint32_t iterations,
        bool trace_paths,
        Options const & opt);

template void TraceRaysCu<float>::trace_rays_cu<float>(
        std::vector<pos_t> const & start_position,
        std::vector<float> const & start_direction,
        std::vector<pos_t> & end_position,
        std::vector<float> & end_direction,
        std::vector<brightness_t> & remaining_light,
        std::vector<pos_t> & path,
        std::vector<float> const & invscale_vec,
        brightness_t minimum_brightness,
        uint32_t iterations,
        bool trace_paths,
        Options const & opt);

